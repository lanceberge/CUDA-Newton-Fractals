#include "hip/hip_runtime.h"
#include "complex.h"

__host__ __device__ Complex::Complex(dfloat x, dfloat y) : Re(x), Im(y) {}

// return the product of two complex numbers
__host__ __device__ Complex Complex::operator*(const Complex& z2)
{
    dfloat x = Re*z2.Re - Im*z2.Im;
    dfloat y = Re*z2.Im + Im*z2.Re;

    return Complex(x, y);
}

// subtract two complex numbers
__device__ Complex Complex::operator-(const Complex& z2)
{
    return Complex(Re - z2.Re, Im - z2.Im);
}

// divide two complex numbers - implementation from
// https://pixel.ecn.purdue.edu:8443/purpl/WSJ/projects/DirectionalStippling/include/hip/hip_complex.h
__device__ Complex Complex::operator/(const Complex& z2)
{
    dfloat s = (fabs(z2.Re)) + (fabs(z2.Im));
    dfloat oos = 1.0 / s;
    dfloat ars = Re * oos;
    dfloat ais = Im * oos;
    dfloat brs = z2.Re * oos;
    dfloat bis = z2.Im * oos;
    s = (brs * brs) + (bis * bis);
    oos = 1.0 / s;

    dfloat Re = ((ars * brs) + (ais * bis)) * oos;
    dfloat Im = ((ais * brs) - (ars * bis)) * oos;

    return Complex(Re, Im);
}

// print a complex number
void Complex::printComplex(const Complex& z)
{
    printf("Re: %f, Im: %f\n", z.Re, z.Im);
}
