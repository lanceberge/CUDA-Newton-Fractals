#include "hip/hip_runtime.h"
#include "polynomial.h"
#include "complex.h"

// pre: coeffs is dynamically allocated with new
// post: c_coeffs is allocated with hipMalloc
Polynomial::Polynomial(int order, dfloat *coeffs)
{
    this->order = order;
    h_coeffs = coeffs;

    hipMalloc(&c_coeffs, (order+1)*sizeof(dfloat));

    // copy host coefficients to device array
    hipMemcpy(c_coeffs, h_coeffs, (order+1)*sizeof(dfloat), hipMemcpyHostToDevice);
}

// find the first derivative of a polynomial
Polynomial Polynomial::derivative()
{
    dfloat *derivative_coeffs = new dfloat[order];

    // update Pprime coeffs
    for (int i = 0; i < order; ++i) {
        derivative_coeffs[i] = h_coeffs[i]*(order-i);
    }

    return Polynomial(order - 1, derivative_coeffs);
}

// find P(z) - plug in a point z to the polynomial - on host coeffs
__host__ Complex Polynomial::h_Pz(const Complex& z) const
{
    dfloat ReSum = h_coeffs[order];
    dfloat ImSum = 0;

    // zPow on first iteration, then zPow^2, then ^3, etc.
    Complex zPow(z.Re, z.Im);

    // for A, B, C, D in coeffs. of P, return the cumulative sum of Az^4 + Bz^3 + ...
    for (int i = order-1; i >= 0; --i) {
        int coeff = h_coeffs[i];

        // zPow = z, then z^2, then z^3, etc.
        ReSum += coeff*zPow.Re;
        ImSum += coeff*zPow.Im;

        // update zPow to zPow*zPow
        zPow = zPow*z;
    }

    return Complex(ReSum, ImSum);
}

// find P(z) - plug in a point z to the polynomial - on device coeffs
__device__ Complex Polynomial::c_Pz(const Complex& z) const
{
    dfloat ReSum = c_coeffs[order];
    dfloat ImSum = 0;

    // zPow on first iteration, then zPow^2, then ^3, etc.
    Complex zPow(z.Re, z.Im);

    // for A, B, C, D in coeffs. of P, return the cumulative sum of Az^4 + Bz^3 + ...
    for (int i = order-1; i >= 0; --i) {
        int coeff = c_coeffs[i];

        // zPow = z, then z^2, then z^3, etc.
        ReSum += coeff*zPow.Re;
        ImSum += coeff*zPow.Im;

        // update zPow to zPow*zPow
        zPow = zPow*z;
    }

    return Complex(ReSum, ImSum);
}

// free memory
__host__ __device__ Polynomial::~Polynomial()
{
    #if !defined(__HIPCC__)
    delete[] h_coeffs;
    hipFree(c_coeffs);
    #endif
}

// the coefficients of a random polynomial - coefficients are
// random between -max and max. seed is the seed for drand
dfloat *randomCoeffs(int order, int max, int seed)
{
    srand48(seed);

    dfloat *coeffs = new dfloat[order + 1];

    for (int i = 0; i < order + 1; ++i) {
        coeffs[i] = -max + 2*max*(drand48());
    }

    return coeffs;
}
