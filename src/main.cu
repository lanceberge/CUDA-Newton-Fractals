#include "hip/hip_runtime.h"
#include "newton.h"
#include <png.h>
#include <stdlib.h>
#include <string>

// Convert values from 0-11 into RGB triplets in ptr
void setRGB(png_byte *ptr, int val)
{
    // arrays for red, green, and blue percentages
    float r[12] = {0    , 0.85 , 0.494, 0.466, 0.635, 0.301, 0.929, 1, .69 , 1   , 0  , 0};
    float g[12] = {0.447, 0.325, 0.184, 0.674, 0.078, 0.745, 0.694, 0, 0.61, 0.75, 0.6, 0.5};
    float b[12] = {0.741, 0.098, 0.556, 0.188, 0.184, 0.933, 0.125, 0, 0.85, 0.8 , 0.3, 0.5};

    // convert into RGB triplets by multiplying each by 256
    ptr[0] = (int)(r[val]*256);
    ptr[1] = (int)(g[val]*256);
    ptr[2] = (int)(b[val]*256);
}

// Output the data to png
// Essentially taken from: http://www.labbookpages.co.uk/software/imgProc/files/libPNG/makePNG.c
void writeImage(const char *filename, int width, int height, int *buffer)
{
    FILE *fp = fopen(filename, "wb");

    // initialize some pointers
    png_structp png_ptr = NULL;
    png_infop info_ptr  = NULL;
    png_bytep row       = NULL;

    // set up png and info ptr
    png_ptr  = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    info_ptr = png_create_info_struct(png_ptr);

    setjmp(png_jmpbuf(png_ptr));

    png_init_io(png_ptr, fp);

    // set some metadata
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGB,
                 PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

    png_write_info(png_ptr, info_ptr);

    // write to png row by row
    row = (png_bytep)malloc(3 * width * sizeof(png_byte));

    int x, y;
    for (y = 0; y < height; ++y) {
        for (x = 0; x < height; ++x) {
            setRGB(&(row[x * 3]), buffer[y * width + x]);
        }
        png_write_row(png_ptr, row);
    }

    png_write_end(png_ptr, NULL);

    fclose(fp);
    png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
    png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
    free(row);
}

// perform the iteration and output to png
int main(int argc, char **argv)
{
    if (argc < 2) {
        printf("Example Usage: ./bin/newton <testName> [NRe=300] NIm=300] [ReSpacing=3] [ImSpacing=3] \
                [L1=false] [step=false] \n");

        printf("testName  - name of the test, if bigTest or bigTest2, the other options will be ignored\n");
        printf("NRe       - Number of real points to run iteration on\n");
        printf("NIm       - number of imaginary points to run iteration on\n");
        printf("ReSpacing - if 4, then the real values will be spaced from -4 to 4\n");
        printf("ImSpacing - same as ReSpacing but for the imaginary values\n");
        printf("L1        - true if you want to use L1 norm to measure distance\n");
        printf("step      - true if you want to output a png for each step\n");
        exit(-1);
    }

    // device array pointers
    int     *closest;
    Complex *solns;
    Complex *zValsInitial;
    Complex *zVals;

    // will be initialized below based on which test we use
    int NRe          = 300;
    int NIm          = 300;
    dfloat ReSpacing = 3;
    dfloat ImSpacing = 3;
    int norm         = 2;
    bool step        = false;

    int order;
    Polynomial P;

    char *testName = argv[1];

    // test on -4x^3 + 6x^2 + 2x = 0, which has roots
    // 0, ~1.78, ~-.28
    if (strcmp(testName, "smallTest") == 0) {
        order = 3;

        // create a polynomial
        dfloat *coeffs = new dfloat[4]{-4, 6, 2, 0};
        P.coeffs = coeffs;
        P.order = order;

        // the spacing on our grid, i.e. 1000 => run iteration on Nx and Ny evenly
        // spaced points from -1000 to 1000 on x and y
        ReSpacing = 4;
        ImSpacing = 4;
    }

    // random polynomial of order 7
    else if (strcmp(testName, "bigTest") == 0) {
        int max = 10;
        int seed = 123456;
        order = 7;

        // create a random order 7 polynomial
        P = randomPolynomial(order, max, seed);

        ReSpacing = 4;
        ImSpacing = 4;
    }

    // order 12
    else if (strcmp(testName, "bigTest2") == 0) {
        // create a random order 11 polynomial
        int max = 50;
        int seed = 654321;

        order = 12;

        ReSpacing = 5;
        ImSpacing = 5;
        P = randomPolynomial(order, max, seed);
    }

    else {
        for (int i = 2; i < argc; ++i) {

            // the value to set - i.e. NRe, L1, step
            char *token = strtok(argv[i], "=");

            // what to set it to
            char *val = strtok(NULL, "=");

            if (val != NULL)
            {

                if (strcmp(token, "NRe") == 0)
                    // if nothing is specified, set to 3, else to the specified value
                    NRe = atoi(val);

                else if (strcmp(token, "NIm") == 0)
                    NIm = atoi(val);

                else if (strcmp(token, "ReSpacing") == 0)
                    ReSpacing = atoi(val);

                else if (strcmp(token, "ImSpacing") == 0)
                    ImSpacing = atoi(val);

            }
        }

        // TODO prompt to enter a polynomial
    }

    // set step and L1, same as above - needs to be done regardless of the test
    for (int i = 2; i < argc; ++i) {
        // the value to set - i.e. NRe, L1, step
        char *token = strtok(argv[i], "=");

        // what to set it to
        char *val = strtok(NULL, "=");

        if (strcmp(token, "L1") == 0)
            norm = strcmp(val, "true") == 0 ? 1 : 2;

        else if (strcmp(token, "step") == 0)
            step = strcmp(val, "true") == 0 ? true : false;
    }

    // P' - derivative of P
    Polynomial Pprime = derivative(P);

    // device versions for newtonIterate
    Polynomial c_P      = deviceP(P);
    Polynomial c_Pprime = deviceP(Pprime);

    int N   = NRe * NIm;

    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1) / 16, (NRe + 16 - 1) / 16);

    // arrays for initial points and points following iteration
    hipMalloc(&zValsInitial, N * sizeof(Complex));
    hipMalloc(&zVals       , N * sizeof(Complex));

    Complex *h_zValsInitial = (Complex *)malloc(N * sizeof(Complex));
    Complex *h_zVals        = (Complex *)malloc(N * sizeof(Complex));

    // initialize arrays - evenly spaced over complex plane
    fillArrays<<<G, B>>>(ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

    // copy to host
    hipMemcpy(h_zValsInitial, zValsInitial, N * sizeof(Complex), hipMemcpyDeviceToHost);

    // perform 500 steps of the iteration and copy result back to host
    newtonIterate<<<G, B>>>(zVals, c_P, c_Pprime, NRe, NIm, 500);

    // copy result to host
    hipMemcpy(h_zVals, zVals, N * sizeof(Complex), hipMemcpyDeviceToHost);

    // find the solutions - unique values in zVals
    Complex *h_solns = (Complex *)malloc(order * sizeof(Complex));
    int nSolns = findSolns(P, h_solns, h_zVals, order, N);

    // find closest solutions to each point in zVals
    hipMalloc(&closest, N * sizeof(int));

    // copy h_solns to device for use in findClosestSoln
    hipMalloc(&solns, nSolns * sizeof(Complex));
    hipMemcpy(solns, h_solns, nSolns * sizeof(Complex), hipMemcpyHostToDevice);

    if (step) {
        // reset zVals
        fillArrays<<<G, B>>>(ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

        for (int i = 0; i < 50; ++i) {
            // perform one iteration, copy back to host, then output image
            hipMemcpy(h_zVals, zVals, N * sizeof(Complex), hipMemcpyDeviceToHost);

            // find the closest solution to each value in zVals and store it in closest
            findClosestSoln<<<G, B>>>(closest, zVals, NRe, NIm, solns, nSolns, norm);

            // fill *closest with an integer corresponding to the solution its closest to
            // i.e. 0 for if this point is closest to solns[0]
            int *h_closest = (int *)malloc(N * sizeof(int));

            // copy results back to host
            hipMemcpy(h_closest, closest, N * sizeof(int), hipMemcpyDeviceToHost);

            // output image
            writeImage(("plots/"+std::string(testName)+"Step-"+std::to_string(i)+".png").c_str(),
                       NRe, NIm, h_closest);

            newtonIterate<<<G, B>>>(zVals, c_P, c_Pprime, NRe, NIm, 1);
        }
    }

    // find the closest solution to each value in zVals and store it in closest
    findClosestSoln<<<G, B>>>(closest, zVals, NRe, NIm, solns, nSolns, norm);

    // fill *closest with an integer corresponding to the solution its closest to
    // i.e. 0 for if this point is closest to solns[0]
    int *h_closest = (int *)malloc(N * sizeof(int));

    // copy results back to host
    hipMemcpy(h_closest, closest, N * sizeof(int), hipMemcpyDeviceToHost);

    // output image
    writeImage(("plots/"+std::string(testName)+".png").c_str(), NRe, NIm, h_closest);

    // free heap memory
    hipFree(closest)        ; free(h_closest)     ;
    hipFree(zVals)          ; free(h_zVals)       ;
    hipFree(zValsInitial)   ; free(h_zValsInitial);
    hipFree(c_P.coeffs)     ; free(P.coeffs)      ;
    hipFree(c_Pprime.coeffs);
    hipFree(solns)          ; free(h_solns)       ;
    return 0;
}
