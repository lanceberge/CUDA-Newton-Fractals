#include "hip/hip_runtime.h"
#include "newton.h"
#include <string>
#include <stdlib.h>

static int NRe;
static int NIm;

// perform Nits iterations and copy result back to host
void iterate(Polynomial c_P, Polynomial c_Pprime, int Nits, Complex *zVals, Complex *h_zVals)
{
    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NIm + 16 - 1)/16);

    // then perform the newton iteration and copy result back to host
    newtonIterate <<< G, B >>> (zVals, c_P, c_Pprime, NRe, NIm, Nits);

    // copy result to host
    hipMemcpy(h_zVals, zVals, NRe*NIm*sizeof(Complex), hipMemcpyDeviceToHost);
}

// find the solutions given h_zVals and output to CSV
void outputSolns(Polynomial P, Complex *h_zVals, Complex **h_solns, int nSolns, int N, std::string filename)
{
    // total number of points
    // find the solutions to this polynomial - the unique points in zVals
    *h_solns = (Complex *)malloc(nSolns * sizeof(Complex));
    nSolns = findSolns(P, *h_solns, h_zVals, nSolns, N);

    std::string solnFilename = "data/"+filename+"Solns.csv";

    FILE *fp = fopen(solnFilename.c_str(), "w");

    // print our header
    fprintf(fp, "Re, Im\n");

    Complex *solns = *h_solns;

    for (int i = 0; i < nSolns; ++i)
    {
        fprintf(fp, "%f, %f\n", solns[i].Re, solns[i].Im);
    }

    fclose(fp);
}

// find the solutions each val in h_zVals is closest to, then output the index of that solution,
// along with the corresponding initial point to a CSV
void outputVals(Complex *zVals, Complex *h_zVals, Complex *h_solns, Complex *h_zValsInitial,
                int nSolns, std::string filename, int norm, int step=-1)
{
    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);

    int *closest;
    hipMalloc(&closest, NRe*NIm*sizeof(int));

    Complex *solns;
    hipMalloc(&solns, nSolns*sizeof(Complex));
    hipMemcpy(solns, h_solns, nSolns*sizeof(Complex), hipMemcpyHostToDevice);

    findClosestSoln <<< G, B >>> (closest, zVals, NRe, NIm, solns, nSolns, norm);

    // fill *closest with an integer corresponding to the solution its closest to
    // i.e. 0 for if this point is closest to solns[0]
    int *h_closest = (int *)malloc(NRe*NIm * sizeof(int));

    // copy results back to host
    hipMemcpy(h_closest, closest, NRe*NIm*sizeof(int), hipMemcpyDeviceToHost);

    // output data and solutions to CSVs
    std::string outputFilename;

    if (step == -1)
        outputFilename = "data/"+filename+"Data.csv";

    else
        outputFilename = "data/"+filename+"Data-"+std::to_string(step)+".csv";

    FILE *fp = fopen(outputFilename.c_str(), "w");

    for (int i = 0; i < NRe*NIm; ++i)
        fprintf(fp, "%f, %f, %d\n", h_zValsInitial[i].Re, h_zValsInitial[i].Im, h_closest[i]);

    fclose(fp);

    hipFree(closest); free(h_closest);
    hipFree(solns);
}

int main(int argc, char **argv)
{
    if (argc < 4)
    {
        printf("Usage: ./newton <NRe> <NIm> <Test> [step]\n");
        printf("NRe  - Number of real points to run iteration on\n");
        printf("NIm  - number of imaginary points to run iteration on\n");
        printf("Test - Which test to run\n");
        printf("Step - optional, use to output at each step\n");
        exit(-1);
    }

    char *test = argv[3];

    Polynomial P;

    Complex *zValsInitial;
    Complex *zVals;
    int order;

    dfloat ReSpacing;
    dfloat ImSpacing;

    // test on -4x^3 + 6x^2 + 2x = 0, which has roots
    // 0, ~1.78, ~-.28
    if (strcmp(test, "smallTest") == 0 || strcmp(test, "smallTestL1") == 0)
    {
        order = 3;

        // create a polynomial
        dfloat *coeffs = new dfloat[4] {-4, 6, 2, 0};
        P.coeffs = coeffs;
        P.order = order;

        // the spacing on our grid, i.e. 1000 => run iteration on Nx and Ny evenly
        // spaced points from -1000 to 1000 on x and y
        ReSpacing = 4;
        ImSpacing = 4;
    }

    // random polynomial of order 7
    else if (strcmp(test, "bigTest") == 0 || strcmp(test, "bigTestL1") == 0)
    {
        int max = 10;
        int seed = 123456;
        order = 7;

        // create a random order 7 polynomial
        P = randomPolynomial(order, max, seed);

        ReSpacing = 4;
        ImSpacing = 4;
    }

    // order 12
    else if (strcmp(test, "bigTest2") == 0 || strcmp(test, "bigTest2L1") == 0)
    {
        // create a random order 11 polynomial
        int max = 50;
        int seed = 654321;

        order = 12;

        ReSpacing = 15;
        ImSpacing = 5;
        P = randomPolynomial(order, max, seed);
    }

    else if (strcmp(test, "bigTest3") == 0 || strcmp(test, "bigTest3L1") == 0)
    {
        // create a random order 11 polynomial
        int max = 100;
        int seed = 123456;

        order = 50;

        ReSpacing = 50;
        ImSpacing = 25;
        P = randomPolynomial(order, max, seed);
    }

    else
        return 0;

    // P' - derivative of P
    Polynomial Pprime = derivative(P);

    // device versions for newtonIterate
    Polynomial c_P      = deviceP(P);
    Polynomial c_Pprime = deviceP(Pprime);

    Complex *h_solns = (Complex *)malloc(order*sizeof(Complex));

    NRe = atoi(argv[1]);
    NIm = atoi(argv[2]);

    int N = NRe*NIm;

    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);

    // arrays for initial points and points following iteration
    hipMalloc(&zValsInitial, N*sizeof(Complex));
    hipMalloc(&zVals,        N*sizeof(Complex));

    Complex *h_zValsInitial = (Complex *)malloc(N*sizeof(Complex));
    Complex *h_zVals        = (Complex *)malloc(N*sizeof(Complex));

    fillArrays <<< G, B >>> (ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

    hipMemcpy(h_zValsInitial, zValsInitial, N*sizeof(Complex), hipMemcpyDeviceToHost);

    // perform 500 steps of the iteration and copy result back to host
    iterate(c_P, c_Pprime, 500, zVals, h_zVals);
    hipMemcpy(h_zVals, zVals, N*sizeof(Complex), hipMemcpyDeviceToHost);

    // find the solutions and output to CSV
    outputSolns(P, h_zVals, &h_solns, order, N, test);

    int norm;

    // output solutions to file and store them
    if (argc > 4 && strcmp(argv[4], "L1") == 0 ||
        argc > 5 && strcmp(argv[5], "L1") == 0)
        norm = 1;

    else
        norm = 2;

    if (argc > 4 && strcmp(argv[4], "step") == 0) {

        for (int i = 0; i < 100; ++i)
        {
            // output then perform 1 iteration
            outputVals(zVals, h_zVals, h_solns, h_zValsInitial, order, test, norm, i);
            iterate(c_P, c_Pprime, 1, zVals, h_zVals);
        }
    }

    else
    {
        // otherwise just do one output
        outputVals(zVals, h_zVals, h_solns, h_zValsInitial, order, test, norm);
    }


    hipFree(zVals)          ; free(h_zVals)       ;
    hipFree(zValsInitial)   ; free(h_zValsInitial);
    hipFree(c_P.coeffs)     ; free(P.coeffs)      ;
    hipFree(c_Pprime.coeffs);

    free(h_solns);
    return 0;
}
