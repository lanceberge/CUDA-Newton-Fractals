#include "newton.h"
#include <string>

static int NRe;
static int NIm;

void performIteration(Polynomial P, int ReSpacing, int ImSpacing,
                      std::string filename, int Nits);

int main(int argc, char **argv)
{
    if (argc < 4)
    {
        printf("Usage: ./newton {NRe} {NIm} {Test}\n");
        printf("NRe - Number of real points to run iteration on\n");
        printf("NIm - number of imaginary points to run iteration on\n");
        printf("Test - Which test to run\n");
        exit(-1);
    }

    NRe        = atoi(argv[1]);
    NIm        = atoi(argv[2]);
    char *test = argv[3];

    Polynomial P;

    // test on -4x^3 + 6x^2 + 2x = 0, which has roots
    // 0, ~1.78, ~-.28
    if (strcmp(test, "smallTest") == 0)
    {
        int order = 3;

        // create a polynomial
        dfloat *coeffs = new dfloat[4] {-4, 6, 2, 0};
        P.coeffs = coeffs;
        P.order = order;

        // the spacing on our grid, i.e. 1000 => run iteration on Nx and Ny evenly
        // spaced points from -1000 to 1000 on x and y
        dfloat ReSpacing = 4;
        dfloat ImSpacing = 4;

        int Nits = 100;

        performIteration(P, ReSpacing, ImSpacing, "smallTest", Nits);
    }

    else if (strcmp(test, "bigTest") == 0)
    {
        // create a random order 7 polynomial
        srand48(123456);

        int order = 7;
        dfloat *coeffs = (dfloat *)malloc((order + 1)*sizeof(dfloat));

        for (int i = 0; i < order + 1; ++i)
        {
            coeffs[i] = -10 + 20*(drand48());
        }

        P.coeffs = coeffs;
        P.order = order;

        dfloat ReSpacing = 4;
        dfloat ImSpacing = 4;

        int Nits = 100;

        performIteration(P, ReSpacing, ImSpacing, "bigTest", Nits);
    }

    return 0;
}

void performIteration(Polynomial P, int ReSpacing, int ImSpacing,
                      std::string filename, int Nits)
{
    // total number of points
    int N = NRe*NIm;

    // device arrays
    Complex *zValsInitial;
    Complex *zVals;
    Complex *solns;
    int *closest;

    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);

    // P' - derivative of P
    Polynomial Pprime = derivative(P);

    // device versions for newtonIterate
    Polynomial c_P      = deviceP(P);
    Polynomial c_Pprime = deviceP(Pprime);

    // arrays for solutions
    int order = P.order;

    // arrays for initial points and points following iteration
    hipMalloc(&zValsInitial, N*sizeof(Complex));
    hipMalloc(&zVals,        N*sizeof(Complex));

    fillArrays <<< G, B >>> (ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

    // then perform the newton iteration and copy result back to host
    newtonIterate <<< G, B >>> (zVals, c_P, c_Pprime, NRe, NIm, Nits);

    Complex *h_zValsInitial = (Complex *)malloc(N*sizeof(Complex));
    Complex *h_zVals        = (Complex *)malloc(N*sizeof(Complex));
    hipMemcpy(h_zValsInitial, zValsInitial, N*sizeof(Complex), hipMemcpyDeviceToHost);
    hipMemcpy(h_zVals,        zVals,        N*sizeof(Complex), hipMemcpyDeviceToHost);

    // find the solutions to this polynomial - the unique points in zVals
    hipMalloc(&solns, order*sizeof(Complex));
    Complex *h_solns = (Complex *)malloc(order * sizeof(Complex));
    int nSolns = findSolns(h_solns, h_zVals, order, N);

    // copy to device
    hipMemcpy(solns, h_solns, nSolns*sizeof(Complex), hipMemcpyHostToDevice);

    // fill *closest with an integer corresponding to the solution its closest to
    // i.e. 0 for if this point is closest to solns[0]
    int *h_closest = (int *)malloc(N * sizeof(int));
    hipMalloc(&closest, N*sizeof(int));

    findClosestSoln <<< G, B >>> (closest, zVals, NRe, NIm, solns, nSolns);

    // copy results back to host
    hipMemcpy(h_closest, closest, N*sizeof(int), hipMemcpyDeviceToHost);

    // output data and solutions to CSVs
    std::string outputFilename = "data/"+filename+"Data.csv";
    std::string solnFilename   = "data/"+filename+"Solns.csv";

    outputToCSV(outputFilename.c_str(), N, h_zValsInitial, h_closest);
    outputSolnsToCSV(solnFilename.c_str(), nSolns, h_solns);

    // free memory
    hipFree(zVals)          ; free(h_zVals)       ;
    hipFree(zValsInitial)   ; free(h_zValsInitial);
    hipFree(c_P.coeffs)     ; free(P.coeffs)      ;
    hipFree(c_Pprime.coeffs); free(Pprime.coeffs) ;
    hipFree(closest)        ; free(h_closest)     ;
    hipFree(solns)          ; free(h_solns)       ;
}
