#include "newton.h"
#include <string>
#include <stdlib.h>

static int NRe;
static int NIm;
static int ReSpacing;
static int ImSpacing;

void iterate(Polynomial c_P, Polynomial c_Pprime, int Nits, Complex *zVals, Complex *h_zVals);

void outputSolns(Complex *h_zVals, Complex *h_zValsInitial,
                 Complex **h_solns, int nSolns, std::string filename);

void outputVals(Complex *zVals, Complex *h_zVals, Complex *h_solns, Complex *h_zValsInitial,
                int nSolns, std::string filename, int step=-1);

int main(int argc, char **argv)
{
    if (argc < 4)
    {
        printf("Usage: ./newton <NRe> <NIm> <Test> [step]\n");
        printf("NRe  - Number of real points to run iteration on\n");
        printf("NIm  - number of imaginary points to run iteration on\n");
        printf("Test - Which test to run\n");
        printf("Step - optional, use to output at each step\n");
        exit(-1);
    }

    NRe        = atoi(argv[1]);
    NIm        = atoi(argv[2]);
    char *test = argv[3];

    int N = NRe*NIm;

    Polynomial P;

    Complex *zValsInitial;
    Complex *zVals;
    int order;

    // arrays for initial points and points following iteration
    hipMalloc(&zValsInitial, N*sizeof(Complex));
    hipMalloc(&zVals,        N*sizeof(Complex));

    Complex *h_zValsInitial = (Complex *)malloc(N*sizeof(Complex));
    Complex *h_zVals        = (Complex *)malloc(N*sizeof(Complex));

    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);

    // test on -4x^3 + 6x^2 + 2x = 0, which has roots
    // 0, ~1.78, ~-.28
    if (strcmp(test, "smallTest") == 0)
    {
        order = 3;

        // create a polynomial
        dfloat *coeffs = new dfloat[4] {-4, 6, 2, 0};
        P.coeffs = coeffs;
        P.order = order;

        // the spacing on our grid, i.e. 1000 => run iteration on Nx and Ny evenly
        // spaced points from -1000 to 1000 on x and y
        ReSpacing = 4;
        ImSpacing = 4;
    }

    else if (strcmp(test, "bigTest") == 0)
    {
        // create a random order 7 polynomial
        srand48(123456);

        order = 7;
        dfloat *coeffs = (dfloat *)malloc((order + 1)*sizeof(dfloat));

        for (int i = 0; i < order + 1; ++i)
        {
            coeffs[i] = -10 + 20*(drand48());
        }

        P.coeffs = coeffs;
        P.order = order;

        ReSpacing = 4;
        ImSpacing = 4;
    }

    else
    {
        return 0;
    }

    // P' - derivative of P
    Polynomial Pprime = derivative(P);

    // device versions for newtonIterate
    Polynomial c_P      = deviceP(P);
    Polynomial c_Pprime = deviceP(Pprime);

    Complex *h_solns = (Complex *)malloc(order*sizeof(Complex));

    fillArrays <<< G, B >>> (ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

    hipMemcpy(h_zValsInitial, zValsInitial, N*sizeof(Complex), hipMemcpyDeviceToHost);


    // perform 100 iterations then output solutions
    iterate(c_P, c_Pprime, 100, zVals, h_zVals);

    // output solutions to file and store them
    outputSolns(h_zVals, h_zValsInitial, &h_solns, order, test);
    outputVals(zVals, h_zVals, h_solns, h_zValsInitial, order, test);

    if (argc >= 5 && strcmp(argv[4], "step") == 0)
    {
        // reset arrays
        fillArrays <<< G, B >>> (ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

        hipMemcpy(h_zVals, zVals, N*sizeof(Complex), hipMemcpyDeviceToHost);

        for (int i = 0; i < 100; ++i)
        {
            // output then perform 1 iteration
            outputVals(zVals, h_zVals, h_solns, h_zValsInitial, order, test, i);
            iterate(c_P, c_Pprime, 1, zVals, h_zVals);
        }
    }


    hipFree(zVals)          ; free(h_zVals)       ;
    hipFree(zValsInitial)   ; free(h_zValsInitial);
    hipFree(c_P.coeffs)     ; free(P.coeffs)      ;
    hipFree(c_Pprime.coeffs);

    free(h_solns);
    return 0;
}

void iterate(Polynomial c_P, Polynomial c_Pprime, int Nits, Complex *zVals, Complex *h_zVals)
{
    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);

    // then perform the newton iteration and copy result back to host
    newtonIterate <<< G, B >>> (zVals, c_P, c_Pprime, NRe, NIm, Nits);

    // copy result to host
    hipMemcpy(h_zVals, zVals, NRe*NIm*sizeof(Complex), hipMemcpyDeviceToHost);
}

void outputSolns(Complex *h_zVals, Complex *h_zValsInitial,
                 Complex **h_solns, int nSolns, std::string filename)
{
    // total number of points
    // find the solutions to this polynomial - the unique points in zVals
    *h_solns = (Complex *)malloc(nSolns * sizeof(Complex));
    nSolns = findSolns(*h_solns, h_zVals, nSolns, NRe*NIm);

    std::string solnFilename   = "data/"+filename+"Solns.csv";

    outputSolnsToCSV(solnFilename.c_str(), nSolns, *h_solns);
}

void outputVals(Complex *zVals, Complex *h_zVals, Complex *h_solns, Complex *h_zValsInitial,
                int nSolns, std::string filename, int step)
{
    dim3 B(16, 16, 1);
    dim3 G((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);

    int *closest;
    hipMalloc(&closest, NRe*NIm*sizeof(int));

    Complex *solns;
    hipMalloc(&solns, nSolns*sizeof(Complex));
    hipMemcpy(solns, h_solns, nSolns*sizeof(Complex), hipMemcpyHostToDevice);

    findClosestSoln <<< G, B >>> (closest, zVals, NRe, NIm, solns, nSolns);

    // fill *closest with an integer corresponding to the solution its closest to
    // i.e. 0 for if this point is closest to solns[0]
    int *h_closest = (int *)malloc(NRe*NIm * sizeof(int));

    // copy results back to host
    hipMemcpy(h_closest, closest, NRe*NIm*sizeof(int), hipMemcpyDeviceToHost);

    // output data and solutions to CSVs
    std::string outputFilename;
    if (step == -1)
        outputFilename = "data/"+filename+"Data.csv";

    else
        outputFilename = "data/"+filename+"Data-"+std::to_string(step)+".csv";


    outputToCSV(outputFilename.c_str(), NRe*NIm, h_zValsInitial, h_closest);

    hipFree(closest); free(h_closest);
    hipFree(solns);
}
