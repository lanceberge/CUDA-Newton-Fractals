#include "newton.h"

int main(int argc, int **argv)
{
    if (argc < 4)
    {
        printf("Usage: ./newton {Nx} {Ny} {Nit}\n");
        printf("NRe - Number of real points to run iteration on\n");
        printf("NIm - number of imaginary points to run iteration on\n");
        printf("Nit - Number of iterations to run\n");
        exit(-1);
    }

    int NRe = atoi(argv[2]);
    int NIm = atoi(argv[3]);
    int Nit = atoi(argv[4]);

    // the spacing on our grid, i.e. 1000 => run iteration on Nx and Ny evenly
    // spaced points from -1000 to 1000 on x and y
    int ReSpacing = 1000;
    int ImSpacing = 500;

    // total number of points
    int N = NRe*NIm;

    // arrays for initial points and points following iteration
    dfloat complex *zValsInitial, zVals;
    hipMalloc(&zValsInitial, N*sizeof(dfloat complex));
    hipMalloc(&zVals, N*sizeof(dfloat complex));

    /* dfloat complex zValsInitial= (dfloat complex *)malloc(Nx*Ny*sizeof(dfloat complex)); */
    /* dfloat complex *zVals = (dfloat complex *)malloc(Nx*Ny*sizeof(dfloat complex)); */

    int B = 256;
    int G = N + B - 1 / B;

    dim3 B2(16, 16, 1);
    dim3 G2((NRe + 16 - 1)/16, (NRe + 16 - 1)/16);
    fillArrays <<< G2, B2 >>> (ReSpacing, ImSpacing, zValsInitial, zVals, NRe, NIm);

    // TODO find a good polynomial P and test on roots
    // TODO confirm output on host
    newtonIterate <<< G, B>>> (zVals, P, Pprime, N, Nit);
    return 0;
}
