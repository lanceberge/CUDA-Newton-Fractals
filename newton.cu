#include "hip/hip_runtime.h"
#include "newton.h"

// perform an iteration of newton's method with a thread handling
// each point in points
__global__ void newtonIterate(PointChange **points, Polynomial *P, Polynomial *Pprime, int N, int Nit)
{
    int n = threadIdx.x + blockIdx.x + blockDim.x;

    if (n < N)
    {
        // peform Nit iterations of z_i+1 = z_i - P(z_i) / P'(z_i)
        for (int i = 0; i < Nit; ++i)
        {
            /* PointChange *p = points[i]; */
            Point *z = points[i]->after;

            // find P(z) and P'(z)
            Point Pz = Pz(P, z);
            Point Pprimez = Pz(Pprime, z);
        }
    }
}
